#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <ATen/cuda/HIPContext.h>


#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <iostream>
#include <algorithm>
#include <vector>
#include <cmath>

#include "util.cuh"

template <typename scalar_t>
__global__ void paged_attention_forward_kernel(
    const int max_context_len,
    const int dim,
    const float scale,
    scalar_t* __restrict__ Q,       // [length, num_heads, dim]
    scalar_t* __restrict__ K,       // [length, num_heads, dim]
    scalar_t* __restrict__ V,       // [length, num_heads, dim]
    int* __restrict__ offsets, // [length]
    scalar_t* __restrict__ S,
    scalar_t* __restrict__ P,
    scalar_t* __restrict__ O
) {
    const int thread_id     = threadIdx.x;
    const int block_dim     = blockDim.x;
    const int block_id      = blockIdx.x;
    const int head_id       = blockIdx.y;
    const int batch_size    = gridDim.x;
    const int num_heads     = gridDim.y;

    const int beg_idx = (block_id == 0)? 0 : offsets[block_id - 1];
    const int end_idx = offsets[block_id];
    const int size = end_idx - beg_idx;
    // S, P have shape [batch_size, num_heads, max_context_len, max_context_len]
    for(int i = thread_id; i < size; i += block_dim) {
        const int shifted_i = i + beg_idx;
        for(int j = 0; j < size; ++j) {
            const int shifted_j = j + beg_idx;
            int S_idx = (num_heads * max_context_len * max_context_len) * block_id + \
                        (max_context_len * max_context_len) * head_id + \
                        (max_context_len) * i + j;
            if (i >= j)
            {
                for(int k = 0; k < dim; ++k) {
                    int Q_idx = (dim * num_heads) * shifted_i + dim * head_id + k;
                    int K_idx = (dim * num_heads) * shifted_j + dim * head_id + k;
                    S[S_idx] += Q[Q_idx] * K[K_idx] * scale;
                }
            } else {
                S[S_idx] = -10000.0;
            }
        }
    }
    float val_sum;
    const int idx_beg = (num_heads * max_context_len * max_context_len) * block_id + \
                        (max_context_len * max_context_len) * head_id;
    // batch_size, num_heads, max_context_len, max_context_len
    for (int i = 0; i < max_context_len; ++i) {
        val_sum = 1e-9;
        for(int j = thread_id; j < size; j += block_dim) {
            float exp_val = exp(S[idx_beg + max_context_len * i + j]);
            val_sum += exp_val;
        }
        __syncthreads();
        val_sum = blockReduceSum<float>(val_sum);
        for(int j = thread_id; j < size; j += block_dim) {
            float exp_val = exp(S[idx_beg + max_context_len * i + j]);
            P[idx_beg + max_context_len * i + j] = (scalar_t)(exp_val / val_sum);
        }
    }
    for(int i = thread_id; i < max_context_len; i += block_dim) {
        const int shifted_i = beg_idx + i;
        for(int j = 0; j < max_context_len; ++j) {
            const int shifted_j = beg_idx + j;
            for(int k = 0; k < dim; ++k) {
                int P_idx = (num_heads * max_context_len * max_context_len) * block_id + \
                            (max_context_len * max_context_len) * head_id + \
                            (max_context_len) * i + j;
                int V_idx = (num_heads * dim) * shifted_j + dim * head_id + k;
                int O_idx = (num_heads * dim) * shifted_i + dim * head_id + k;
                O[O_idx] += P[P_idx] * V[V_idx];
            }
        }
    }
}

template <typename scalar_t>
__global__ void paged_kv_attention_forward_kernel(
    const int max_context_len,
    const int dim,
    const float scale,
    scalar_t* __restrict__ Q,               // [length, num_heads, dim]
    scalar_t* __restrict__ K,               // [length, num_heads, dim]
    scalar_t* __restrict__ V,               // [length, num_heads, dim]
    scalar_t* __restrict__ K_cache,         // [cache_size, num_heads, dim]
    scalar_t* __restrict__ V_cache,         // [cache_size, num_heads, dim]
    int* __restrict__ cache_indices,        // [length]
    int* __restrict__ offsets,              // [batch_size]
    scalar_t* __restrict__ S,               // [batch_size, num_heads, max_context_len + 1]
    scalar_t* __restrict__ P,               // [batch_size, num_heads, max_context_len + 1]
    scalar_t* __restrict__ O                // [length, num_heads, dim]
) {
    const int thread_id = threadIdx.x;
    const int block_dim = blockDim.x;
    const int block_id = blockIdx.x;
    const int head_id = blockIdx.y;
    const int batch_size = gridDim.x;
    const int num_heads = gridDim.y;

    const int beg_idx = (block_id == 0)? 0 : offsets[block_id - 1];
    const int end_idx = offsets[block_id];
    const int size = end_idx - beg_idx;

     // S[i] = K_cache[i][j] * Q[j];
    printf("block_id %d beg_id %d size %d\n",block_id, beg_idx, size);
    for(int i = thread_id; i < size; i += block_dim) {
        int S_idx = ((1 + max_context_len) * num_heads) * block_id + \
                    (1 + max_context_len) * head_id + i;
        for(int j = 0;j < dim; ++j) {
            int K_cache_idx = (dim) * head_id + (num_heads * dim) * cache_indices[beg_idx + i] + j;
            int Q_idx = (num_heads * dim) * block_id + dim * head_id + j;
            S[S_idx] += K_cache[K_cache_idx] * Q[Q_idx];
        }
    }
    __syncthreads();
    scalar_t tmp = 0.0;
    for(int i = thread_id; i < dim; i += block_dim) {
        int Q_idx = (num_heads * dim) * block_id + dim * head_id + i;
        int K_idx = Q_idx;
        tmp += Q[Q_idx] * K[K_idx];
    }

    // S shape [batch_size, num_heads, max_context_len + 1]
    S[(num_heads * (1 + max_context_len)) * block_id + (max_context_len + 1) * head_id + size] = blockReduceSum<float>(tmp);


    float exp_sum = 0;
    for(int i = thread_id; i < size + 1; i += block_dim) {
        int idx = ((1 + max_context_len) * num_heads) * block_id + (1 + max_context_len) * head_id + i;
        float exp_val = exp(S[idx]);
        exp_sum += exp_val;
    }

    exp_sum = blockReduceSum<float>(exp_sum);
    for(int i = thread_id; i < size + 1; i += block_dim) {
        int idx = ((1 + max_context_len) * num_heads) * block_id + (1 + max_context_len) * head_id + i;
        float exp_val = exp(S[idx]);
        P[idx] = exp_val / exp_sum;
    }

    for (int j = thread_id; j < dim; j += block_dim) {
        for (int i = 0; i < size; ++i) {
            int O_idx = (num_heads * dim) * block_id + (dim) * head_id + j;
            int P_idx = (num_heads * (1 + max_context_len)) * block_id + \
                        ((1 + max_context_len)) * head_id + i;
            int V_idx = dim * head_id + (num_heads * dim) * cache_indices[beg_idx + i] + j;
            O[O_idx] += P[P_idx] * V_cache[V_idx];
        }
    }
    for (int j = thread_id; j < dim; j += block_dim) {
        int O_idx = (num_heads * dim) * block_id + (dim) * head_id + j;
        int P_idx = (num_heads * (1 + max_context_len)) * block_id + \
                    ((1 + max_context_len)) * head_id + size;
        int V_idx = (num_heads * dim) * block_id + (dim) * head_id + j;
        O[O_idx] += P[P_idx] * V[V_idx];
    }

}

std::vector<torch::Tensor> paged_attention_forward(
    torch::Tensor &Q,       // [length, dim]
    torch::Tensor &K,       // [length, dim]
    torch::Tensor &V,       // [length, dim]
    torch::Tensor &offsets,   // [length]
    int num_heads
) {
    // always perform diagonal masking
    CHECK_INPUT(Q); CHECK_INPUT(K); CHECK_INPUT(V);
    auto batch_size = offsets.size(0);
    auto dim = Q.size(1);
    assert(dim % num_heads == 0);
    auto options = torch::TensorOptions().dtype(Q.scalar_type()).device(torch::kCUDA);
    int max_context_len = offsets[0].item<int>();
    for (int i = 1; i < batch_size; ++i) {
        max_context_len = max(max_context_len, (offsets[i] - offsets[i - 1]).item<int>());
    }

    auto S = torch::zeros({batch_size, num_heads, max_context_len, max_context_len}, options);
    auto P = torch::zeros({batch_size, num_heads, max_context_len, max_context_len}, options);
    auto O = torch::zeros_like(V);
    const int threads = std::min(max_context_len, 1024);
    const dim3 blocks(batch_size, num_heads);
    const hipStream_t stream = at::cuda::getCurrentCUDAStream();

    float scale = 1.0 / std::sqrt(float(dim) / num_heads);

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
        Q.scalar_type(),
        "paged_attention_forward_kernel",
        ([&] {
            paged_attention_forward_kernel<<<blocks, threads, 0, stream>>>(
                max_context_len,
                dim / num_heads,
                scale,
                Q.data_ptr<scalar_t>(),
                K.data_ptr<scalar_t>(),
                V.data_ptr<scalar_t>(),
                offsets.data_ptr<int>(),
                S.data_ptr<scalar_t>(),
                P.data_ptr<scalar_t>(),
                O.data_ptr<scalar_t>()
            );
        })
    );
    return {S, P, O};
}

std::vector<torch::Tensor> paged_kv_attention_forward(
    torch::Tensor &Q,                   // [batch_size, dim]
    torch::Tensor &K,                   // [batch_size, dim]
    torch::Tensor &V,                   // [batch_size, dim]
    torch::Tensor &K_cache,             // [num tokens, num_heads, dim]
    torch::Tensor &V_cache,             // [num tokens, num_heads, dim]
    torch::Tensor &cache_indices,       // [num total working indices]
    torch::Tensor &offsets,             // [batch_size]
    int num_heads
) {
    CHECK_INPUT(Q); CHECK_INPUT(K); CHECK_INPUT(V);
    CHECK_INPUT(K_cache); CHECK_INPUT(V_cache);

    auto batch_size = Q.size(0);
    auto dim = Q.size(1);
    assert(dim % num_heads == 0);

    auto options = torch::TensorOptions().dtype(Q.scalar_type()).device(torch::kCUDA);

    auto max_context_len = offsets[0].item<int>();
    for(int i = 1; i < batch_size; ++i) {
        max_context_len = max(max_context_len, (offsets[i] - offsets[i - 1]).item<int>());
    }

    auto S = torch::zeros({batch_size, num_heads, max_context_len + 1}, options);
    auto P = torch::zeros({batch_size, num_heads, max_context_len + 1}, options);
    auto O = torch::zeros_like(Q);

    const int threads = std::min((int) (max_context_len + 1), 1024);
    const dim3 blocks(batch_size, num_heads);
    const hipStream_t stream = at::cuda::getCurrentCUDAStream();

    float scale = 1.0 / std::sqrt(float(dim) / num_heads);

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
        Q.scalar_type(),
        "paged_kv_attention_forward_kernel",
        ([&] {
            paged_kv_attention_forward_kernel<<<blocks, threads, max_context_len, stream>>>(
                max_context_len,
                dim / num_heads,
                scale,
                Q.data_ptr<scalar_t>(),
                K.data_ptr<scalar_t>(),
                V.data_ptr<scalar_t>(),
                K_cache.data_ptr<scalar_t>(),
                V_cache.data_ptr<scalar_t>(),
                cache_indices.data_ptr<int>(),
                offsets.data_ptr<int>(),
                S.data_ptr<scalar_t>(),
                P.data_ptr<scalar_t>(),
                O.data_ptr<scalar_t>()
            );
        })
    );

    return {S, P, O};
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("paged_attention_forward", &paged_attention_forward, "naive attention forward");
    m.def("paged_kv_attention_forward", &paged_kv_attention_forward, "kv forward");
}
